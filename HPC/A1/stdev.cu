
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <cmath>
using namespace std;
using namespace std::chrono;

__global__ void reduce(float *g_idata, float *g_odata){
    extern __shared__ float sdata[];

    //each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s = 1;s < blockDim.x; s *= 2){
        if(tid % (2 * s) == 0){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void compute_difference_between_mean_and_elements(float *difference_array, float *original_array, float mean){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    difference_array[tid] = (original_array[tid] - mean) * (original_array[tid] - mean);
}

void sum_CPU(float *host_input, float *host_output, unsigned int size){
    host_output[0] = 0;
    auto start = high_resolution_clock::now();
    for(int i = 0;i < size;i ++){
        host_output[0] += host_input[i];
    }
    auto stop = high_resolution_clock::now();
    auto time_req = duration_cast<microseconds>(stop - start).count();
    cout << endl << "Time required for CPU : " << time_req << " microseconds "<< endl;
    cout << endl << " Sum from CPU : " << host_output[0] << endl;
}

void compute_sum_cpu(float *cpu_input, float *cpu_output, unsigned int n){
    
    for(unsigned int i = 0;i < n;i ++){
        cpu_output[0] += cpu_input[i];
    }
    
}

int main(){
    
    int maxThreads = 1024;
    
    float *host_input, *host_output, *device_input, *device_output;
    float *cpu_input, *cpu_output;

    int n = 2 << 20;
    size_t size = n * sizeof(int);

    //CPU sum
    cpu_input = (float *)malloc(size);
    cpu_output = (float *)malloc(sizeof(int));
    cpu_output[0] = 0;

    for(unsigned int i = 0;i < n;i ++){
  		cpu_input[i] = rand()%10 ;
    }

    sum_CPU(cpu_input, cpu_output, n);

    host_input = (float *)malloc(size);
    for(int i = 0;i < n;i ++){
        host_input[i] = cpu_input[i];
    }
    
    int blocks = n / maxThreads;
    host_output = (float *)malloc(blocks * sizeof(int));

    const dim3 block_size(maxThreads, 1, 1);
    const dim3 grid_size(blocks, 1, 1);
    
    hipMalloc(&device_input, size);
    hipMalloc(&device_output, blocks * sizeof(int));
    //copy reduce copy and sum

    hipMemcpy(device_input, host_input, size, hipMemcpyHostToDevice);

    reduce<<<grid_size, block_size, maxThreads * sizeof(float)>>>(device_input, device_output);

    hipMemcpy(host_output, device_output, blocks * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 1;i < blocks; i++){
        host_output[0] += host_output[i];
    }

    cout << endl << " Sum from GPU : " << *host_output << endl;
    
    float mean = float(host_output[0] / n);
    cout << endl << " Mean of the array : " << mean << endl;

    //Compute array of [(x1-mean)^2, (x2-mean)^2, (x3-mean)^2, ... ]
    float *array_of_difference_between_mean_and_elements_device;
    hipMalloc(&array_of_difference_between_mean_and_elements_device, size);
    compute_difference_between_mean_and_elements<<<grid_size, block_size>>>(array_of_difference_between_mean_and_elements_device, device_input, mean);

    //Compute (x1-mean)^2 + (x2 - mean) ^ 2 + ...
    float *output_array_for_sum_of_difference_between_elements, *output_array_for_sum_of_difference_between_elements_host;
    output_array_for_sum_of_difference_between_elements_host = (float *)malloc(blocks * sizeof(int));//for host
    hipMalloc(&output_array_for_sum_of_difference_between_elements, blocks * sizeof(int));//for elements
    reduce<<<grid_size, block_size, maxThreads * sizeof(int)>>>(array_of_difference_between_mean_and_elements_device, output_array_for_sum_of_difference_between_elements);
    hipMemcpy(output_array_for_sum_of_difference_between_elements_host, output_array_for_sum_of_difference_between_elements, blocks * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 1;i < blocks;i ++){
        output_array_for_sum_of_difference_between_elements_host[0] += output_array_for_sum_of_difference_between_elements_host[i];
    }
    
    // Compute variance i.e ((x1 - mean)^2 + (x2 - mean)^2 ...) / n
    output_array_for_sum_of_difference_between_elements_host[0] = output_array_for_sum_of_difference_between_elements_host[0] / n;
    cout << endl << "Variance from GPU : " << output_array_for_sum_of_difference_between_elements_host[0] << endl;

    //Compute square root of (x1 - mean) ^ 2 + (x2 - mean) ^ 2 ...
    output_array_for_sum_of_difference_between_elements_host[0] = sqrt(output_array_for_sum_of_difference_between_elements_host[0]);

    cout << endl << "Standard deviation from  GPU : " << output_array_for_sum_of_difference_between_elements_host[0] << endl;

}
