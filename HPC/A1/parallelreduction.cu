
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void minimum(int *input)
{
  int tid = threadIdx.x;
  int step_size = 1;
  int number_of_threads = blockDim.x;

  while(number_of_threads > 0)
  {
    if(tid < number_of_threads)
    {
      int first = tid*step_size*2;
      int second = first + step_size;
      if(input[first] > input[second])
      {
        input[first] = input[second];
      }
    }
    step_size *= 2;
    number_of_threads /=2;
  }
}

__global__ void sum(int *input)
{
  int step_size = 1;
  int number_of_threads = blockDim.x;
  int tid = threadIdx.x;

  while(number_of_threads > 0)
  {
    if(tid < number_of_threads)
    {
      int first = tid*step_size*2;
      int second = first + step_size;
      input[first] += input[second];
    }
    step_size *=2;
    number_of_threads /= 2;
  }
}
int main()
{
  int n;
  cout<<"Enter no of elements"<<"\n";
  cin>>n;

  srand(n);
  int *arr = new int[n];
  for(int i=0;i<n;i++)
  {
    arr[i] = rand();
  }

  for(int i=0;i<n;i++)
  {
    cout<<arr[i]<<" ";
  }
  cout<<"\n";

  int size = n*sizeof(int);
  int *arr_d,result1;

  hipMalloc(&arr_d,size);
  hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);

  minimum<<<1,n/2>>>(arr_d);

  hipMemcpy(&result1,arr_d,sizeof(int),hipMemcpyDeviceToHost);
  cout<<"Minimum Element  = "<<result1;

  hipFree(arr_d);

  int *arr_sum,result2;
  hipMalloc(&arr_sum,size);
  hipMemcpy(arr_sum,arr,size,hipMemcpyHostToDevice);

  sum<<<1,n/2>>>(arr_sum);

  hipMemcpy(&result2,arr_sum,size,hipMemcpyDeviceToHost);
  cout<<"Sum = "<<result2;

  hipFree(arr_sum);
  return 0;
}